#include "hip/hip_runtime.h"

#include "SAX.h"
#include <hip/hip_runtime.h>

//#define THREADS 250
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


uchar* d_collisionTable;
uchar* d_cols;
CudaWord* d_words;


void startEvent(hipEvent_t &start, hipEvent_t &stop){
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
}

float endEvent(hipEvent_t &start, hipEvent_t &stop){
	float elapsedTime;
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );    
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
	return elapsedTime;
}

void devInit(){
	int size0 = NUM_WORDS * NUM_WORDS * sizeof(uchar);
	int size1 = MASK_SIZE * ITERATIONS * sizeof(uchar);
	int size2 = NUM_WORDS * sizeof(CudaWord);

	clock_t start, end;
	start = clock();
	hipMalloc( &d_collisionTable, size0 );
	hipMalloc( &d_cols, size1 );
	hipMalloc( &d_words, size2 );	
	end = clock();
	printf("\tExec time cudaMallocs: %d ms\n", end - start );
	
	start = clock();
	hipMemset( d_collisionTable, 0,  size0 );
	end = clock();
	printf("\tExec time hipMemset: %d ms\n", end - start );

	start = clock();
	hipMemcpy( d_words, h_words, size2, hipMemcpyHostToDevice );
	end = clock();
	printf("\tExec time hipMemcpy: %d ms\n", end - start );
}


void devFree() {
	hipFree( d_collisionTable );
	hipFree( d_cols );
	hipFree( d_words );
}


__global__
void cudaRP_1D_OneIteration(uchar* collisionTable, CudaWord* words, uchar* cols){
	uint index_i = blockIdx.x * blockDim.x + threadIdx.x;	

	if( index_i >= NUM_WORDS ) return;

	uint index_j = blockIdx.y * blockDim.y;

	__shared__ CudaWord _words [ THREADS ];
	__shared__ CudaWord _maskedWords [ THREADS ];

	_maskedWords [ threadIdx.x ] = words [ index_i ];
	_words [ threadIdx.x ] = words [ index_j + threadIdx.x ];

	__syncthreads();	
	
	bool flag;
	for( uint i = 0; i < blockDim.x; i++ )
	{
		if ( index_i != index_j && index_j < NUM_WORDS ) 
		{
			flag = true;
			for (uchar j = 0; j < MASK_SIZE; j++) {
				if ( _words[ threadIdx.x ].values[ cols[j] ] != _maskedWords[ i ].values[ cols[j] ] ) {
					flag = false;
					break;
				}
			}

			if ( flag ) {
				collisionTable[index_j * NUM_WORDS + index_i] += 1;
			}
		}
		index_j++;
	}
}


__global__
void cudaRP_1D_AllIterations(uchar* collisionTable, CudaWord* words, uchar* cols){
	uint index_i = blockIdx.x * blockDim.x + threadIdx.x;	

	if( index_i >= NUM_WORDS ) return;

	uint index_j = blockIdx.y * blockDim.y;

	__shared__ CudaWord _words [ THREADS ];
	__shared__ CudaWord _maskedWords [ THREADS ];

	_maskedWords [ threadIdx.x ] = words [ index_i ];
	_words [ threadIdx.x ] = words [ index_j + threadIdx.x ];

	__syncthreads();	
	
	bool flag;
	uchar count;
	for( uint i = 0; i < blockDim.x; i++ )
	{
		if ( index_i != index_j && index_j < NUM_WORDS ) 
		{
			count = 0;
			for( int iter = 0; iter < ITERATIONS; iter++ )
			{
				flag = true;
				for (uchar j = iter * MASK_SIZE; j < iter * MASK_SIZE + MASK_SIZE; j++) {
					if ( _words[ threadIdx.x ].values[ cols[j] ] != _maskedWords[ i ].values[ cols[j] ] ) {
						flag = false;
						break;
					}
				}

				if ( flag ) {
					count++;
				}
			}

			collisionTable[index_j * NUM_WORDS + index_i] = count;
		}
		index_j++;
	}
}


__global__
void cudaRP_2D_OneIteration(uchar* collisionTable, CudaWord* words, uchar* cols){
	uint index_i = blockIdx.x * blockDim.x + threadIdx.x;
	uint index_j = blockIdx.y * blockDim.y + threadIdx.y;

	if( index_i == index_j || index_i >= NUM_WORDS || index_j >= NUM_WORDS ) return;

	uchar count = 0;
	uint index = index_j * NUM_WORDS + index_i;
// 	CudaWord word_i = words[ index_i ];
// 	CudaWord word_j = words[ index_j ];

	bool flag = true;
	for (uchar c = 0; c < MASK_SIZE; c++) {
//		if ( word_i.values[ cols[c] ] != word_j.values[ cols[c] ] ) {
		if ( words[ index_j ].values[ cols[c] ] != words[ index_i ].values[ cols[c] ] ) {
			flag = false;
			break;
		}
	}

	if ( flag ) {
		collisionTable[index] += 1;
	}
}


__global__
void cudaRP_2D_AllIterations(uchar* collisionTable, CudaWord* words, uchar* cols){
	uint index_i = blockIdx.x * blockDim.x + threadIdx.x;
	uint index_j = blockIdx.y * blockDim.y + threadIdx.y;

	if( index_i == index_j || index_i >= NUM_WORDS || index_j >= NUM_WORDS ) return;

	uchar count = 0;
	uint index = index_j * NUM_WORDS + index_i;
// 	CudaWord word_i = words[ index_i ];
// 	CudaWord word_j = words[ index_j ];

	for( int iter = 0; iter < ITERATIONS; iter++ ){
		bool flag = true;
		for (uchar c = iter * MASK_SIZE; c < iter * MASK_SIZE + MASK_SIZE; c++) {
//			if ( word_i.values[ cols[c] ] != word_j.values[ cols[c] ] ) {
			if ( words[ index_j ].values[ cols[c] ] != words[ index_i ].values[ cols[c] ] ) {
				flag = false;
				break;
			}
		}

		if ( flag ) {
			count++;
		}
	}

	collisionTable[index] = count;
}


void devRandomProjection(){

	clock_t start, end;
	start = clock();
	for (int it = 0; it < ITERATIONS; it++) {
		std::set<int> cols_set;
		for (int i = 0; i < MASK_SIZE; i++) {
			int tentativeColumn = rand() % WORD_SIZE;
			if ( cols_set.find(tentativeColumn) == cols_set.end() )
				cols_set.insert(tentativeColumn);
			else
				i--;
		}
		std::copy( cols_set.begin(), cols_set.end (), &h_cols[ MASK_SIZE * it ] ); 

// 		printf("Mask columns (%d): [", it);
// 		for (int i = 0; i < cols_set.size(); i++) {
// 			printf("%d, ", h_cols[it * MASK_SIZE + i]);
// 		}
// 		printf("]\nWORD LIST SIZE: %d \n", cols_set.size());

		hipMemcpy(d_cols, h_cols, MASK_SIZE * ITERATIONS, hipMemcpyHostToDevice);
	}
	end = clock();
	printf("\tExec time cols generation: %d ms\n", (end - start));
	
	int b = (NUM_WORDS + THREADS - 1) / THREADS;
	dim3 blocks ( b , b );
	dim3 threads ( THREADS , THREADS );
	
	hipEvent_t e_start, e_stop;
	startEvent(e_start, e_stop);
 	for (int i = 0; i < ITERATIONS; i++ ) {	
 		cudaRP_1D_OneIteration<<<blocks , THREADS >>>( d_collisionTable, d_words, &d_cols[i*MASK_SIZE] );
//		cudaRP_2D_OneIteration<<<blocks , threads >>>( d_collisionTable, d_words, &d_cols[i*MASK_SIZE] );
//		cudaRP_2D_AllIterations<<<blocks , threads >>>( d_collisionTable, d_words, d_cols );
//		cudaRP_1D_AllIterations<<<blocks , THREADS >>>( d_collisionTable, d_words, d_cols );
 	}
 	printf("\tExec time #%d cudaRP_1D_OneIteration: %lf ms\n", ITERATIONS, endEvent(e_start, e_stop));
//	printf("\tEXEC TIME cudaRP_2D_AllIterations: %3.1f ms\n", endEvent(e_start, e_stop));
	printf("\tConfiguration: <<< (%d,%d) , (%d,%d) >>>\n", b, b, THREADS, 1 );
	
	startEvent(e_start, e_stop);
	hipMemcpy(h_collisionTable, d_collisionTable, NUM_WORDS * NUM_WORDS * sizeof(uchar), hipMemcpyDeviceToHost);
	printf("\tExec time hipMemcpy collision_table: %3.1f ms\n", endEvent(e_start, e_stop));
}


void show1MotifResult() 
{
	int bestMotifSoFar = 0;
	std::vector<int> bestMotifLocationSoFar;
	for (int i = 0; i < NUM_WORDS; i++) {
		int counter = 0;
		std::multimap<uchar, int, std::greater<uchar> > pointers;
		for(int j = 0; j < NUM_WORDS; j++) {
			int index = i * NUM_WORDS + j;

			uchar count = h_collisionTable[index];
			counter += count; 
			pointers.insert( std::make_pair(count, j) );
		}
		if( counter > bestMotifSoFar ) {
			bestMotifSoFar = counter;
			bestMotifLocationSoFar.clear();

			bestMotifLocationSoFar.push_back(i);
			std::multimap<uchar, int, std::greater<uchar> >::iterator iter = pointers.begin();
			for ( ; iter != pointers.end(); iter++) {
				if (iter->first > 0)
					bestMotifLocationSoFar.push_back(iter->second);
			}
		}
	}
	int topK = 25;
	printf("SIZE: %d\n", NUM_WORDS * NUM_WORDS);
	printf("1-MOTIF:\n ");
	for (int t = 0; t < min(topK, (int)bestMotifLocationSoFar.size()); t++) {
		printf("%d, ",  bestMotifLocationSoFar[t]);
	}
 	printf("\n");
}

void testCudaRandomProjection(){	
	clock_t start = clock();
	std::string train = DATASET;
	SAX::loadData(train);
	clock_t end = clock();
	printf("Exec time loadData(): %d ms\n", (end - start));
	
	start = clock();
	devInit();
	end = clock();
	printf("Exec time devInit(): %d ms\n", (end - start));

	start = clock();
	devRandomProjection();
	end = clock();
	printf("Exec time devRandomProjection(): %d ms\n", (end - start));

	devFree();

	start = clock();
	show1MotifResult();
	end = clock();
	printf("Exec time show1MotifResult(): %d ms\n", (end - start));
}


int main(){
	printf("\nWORDS = %d\n", NUM_WORDS);	
	clock_t begin = clock();
	testCudaRandomProjection();
	clock_t end = clock();
	printf("Total exec time was: %d ms\n\n", end - begin);
	return 0;
}